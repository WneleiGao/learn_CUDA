#include "hip/hip_runtime.h"
#include "/home/ywang15/Documents/cuda/cuda_by_examples/common/book.h"
#include <iostream.h>

#define N 3945384953
#define numThreads 256
#define numBlocks 256

__global__ void kernal(float *a, float *b, float *c){
    __shared__ float cache[numThreads];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cid = threadIdx.x;
    float tmp;
    while (tid < N){
        tmp = a[tid] * b[tid];
        cache[cid] += tmp;
        tid = tid + numBlocks * numThreads;
    }
    __syncThreads();

    int iu = blockDim.x / 2;
    while (iu > 0){
        if (tid < iu){
           cache[tid] = cache[tid] + cache[tid+iu];
        }
        __syncThreads();
        iu = iu / 2;
    }
    if (threadIdx.x == 0){
        c[blockIdx] = cache[0];
    }
}

int main( void ){
    float *a, *b, *cpar, c;
    float *d_a, *d_b, *d_c;

    a = new float[N];
    // a  = (float *)malloc( N*sizeof(float));
    b = new float[N];
    cpar = new float[numBlocks];
    for (int i=0; i<N; i++){
        a[i] = i;
        b[i] = i;
    }
    HANDLE_ERROR( hipMalloc( (void **)&d_a, sizeof(float)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_b, sizeof(float)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_c, sizeof(float)*numBlocks ) );

    kernal<<<numBlocks, numThreads>>>(d_a, d_b, d_c);

    HANDLE_ERROR( hipMemcpy( d_a, a, size(float)*N, hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( d_b, b, size(float)*N, hipMemcpyDeviceToHost ) );
    HANDLE_ERROR( hipMemcpy( cpar, d_c, size(float)*numBlocks, hipMemcpyDeviceToHost ) );

    float r;
    for (i=0; i<numBlocks; i++){
        r = r + cpar[i];
    }
    printf("final result %f\n", r);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete [] a;
    delete [] b;
    delete [] cpar;

    return 0;

}
