#include "hip/hip_runtime.h"
#include "/home/ywang15/Documents/cuda/cuda_by_examples/common/book.h"
#include <iostream>

#define N 100

__global__ void add(int *a, int *b, int *c){
    tid = blockIdx.x;
    c[tid] = a[tid] + b[tid];
}

void main(void){
    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    for (int i=0; i<N; i++){
        a[i] = i+1;
        b[i] = i+1;
    }

    HANDLE_ERROR( hipMalloc( (void **)&d_a, sizeof(int)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_b, sizeof(int)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_c, sizeof(int)*N ) );

    HANDLE_ERROR( hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice) );

    add<<<N, 1>>>(d_a, d_b, d_c);
    HANDLE_ERROR( hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost) );

    for (int i=0; i<N; i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
