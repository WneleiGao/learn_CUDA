#include "hip/hip_runtime.h"
#include "/home/ywang15/Documents/cuda/cuda_by_examples/common/book.h"
#include <iostream>

#define N 876982349

__global__ void add(int *a, int *b, int *c){
    id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < N){
        c[id] = a[id] + b[id];
        id = id + blockDim.x * gridDim.x;
    }
}

void main(void){
    int a[N], b[N], c[N];
    int *d_a, *d_b, *d_c;

    for (int i=0; i<N; i++){
        a[i] = i+1;
        b[i] = 2*(i+1);
    }

    HANDLE_ERROR( hipMalloc( (void **)&d_a, sizeof(int)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_b, sizeof(int)*N ) );
    HANDLE_ERROR( hipMalloc( (void **)&d_c, sizeof(int)*N ) );

    HANDLE_ERROR( hipMemcpy(d_a, a, sizeof(int)*N, hipMemcpyHostToDevice) );
    HANDLE_ERROR( hipMemcpy(d_b, b, sizeof(int)*N, hipMemcpyHostToDevice) );

    add<<<128, 128>>>(d_a, d_b, d_c);
    HANDLE_ERROR( hipMemcpy(c, d_c, sizeof(int)*N, hipMemcpyDeviceToHost) );

    for (int i=0; i<N; i++){
        if (a[i]+b[i] != c[i]){
            printf("%d + %d != %d\n", a[i], b[i], c[i]);
            printf("program failed!\n");
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
