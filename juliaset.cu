#include "hip/hip_runtime.h"
#include "/home/ywang15/Documents/cuda/cuda_by_examples/common/book.h"
#include "/home/ywang15/Documents/cuda/cuda_by_examples/common/cpu_bitmap.h"

#define len 200

struct hipComplex{
    float r;
    float i;

    hipComplex(float a, float b) : r(a), i(b) {}

    __device__ hipComplex operator*(hipComplex &c){
        real = r*a - i*b;
        imag = a*i + r*b;
        return hipComplex(real, imag);
    }

    __device__ cuComplx operator+(hipComplex &c){
        return hipComplex(r+c.r, i+c.i);
    }

    __device__ float magnitude2( void ){
        return r*r + i*i;
    }
}

__device__ int julia(int x, int y){
    const float scale = 1.5;
    const float tmp = (float)(len) / 2.0;
    float ix = scale * (tmp-x) / tmp;
    float iy = scale * (tmp-y) / tmp;

    hipComplex a(ix,iy), c(-0.8, 0.156);
    for (int i=0; i<200; i++){
        a = a*a + c;
    }
    if (a.magnitude2() > 1000.0){
        return 0;
    }
    return 1;
}

__global__ void kernal(unsigned char *ptr){
    int idx = blockIdx.x;
    int idy = blockIdx.y;
    int id  = idx + idy * gridDim.x;
    int value = julia(idx, idy);

    ptr[id*4 + 0] = 255 * value;
    ptr[id*4 + 1] = 0;
    ptr[id*4 + 2] = 0;
    ptr[id*4 + 3] = 255;
}

int main( void ){
    CPUBitMap bitmap(len, len);
    unsigned char *d_bmp;

    HANDLE_ERROR( hipMalloc( (void**) &d_map, bitmap.image_size()  ) );
    dims3 grid(len, len);
    kernal<<<dims, 1>>>(d_bmp);

    HANDLE_ERROR( hipMemcpy( bitmap.get_ptr(), d_map,
                              bitmap.image_size(), hipMemcpyDeviceToHost ) );
    hipFree(d_map);
    bitmap.display_and_exit();
    return 0;
}
