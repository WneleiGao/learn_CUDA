#include "hip/hip_runtime.h"
#define INF 2e10f

struct Sphere{
    float x;
    float y;
    float z;
    float radius;
    float r;
    float g;
    float b;
    __device__ float hit(float ox, float oy, float *n){
        float dx = abs(ox-x);
        float dy = abs(oy-y);
        float d  = sqrt(dx*dx + dy*dy);
        if (d < radius){
           dz = sqrt(radius*radius - d*d);
           *n = dz / radius;
           return dz + z;
        }
        return INF
    }
}
